#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "helpers.h"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
  bool hasHitObject;
  bool hasHitScreen;
  float2 screenUV;
};

//new variables
rtDeclareVariable(int, visualizeScreen, , ) = 0;
rtDeclareVariable(int, useScreenSample, , ) = 0;
rtDeclareVariable(float3, upVec,,);
rtDeclareVariable(float, _cx, ,);
rtDeclareVariable(float, _cy, ,);
rtDeclareVariable(int, useDepthOfField, , ) = 1;

rtDeclareVariable(unsigned int, sampleSize, , )=2;
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );


rtDeclareVariable(float3,        eye, , ) = { 1.0f, 0.0f, 0.0f };
rtDeclareVariable(float3,        U, , )   = { 0.0f, 1.0f, 0.0f };
rtDeclareVariable(float3,        V, , )   = { 0.0f, 0.0f, 1.0f };
rtDeclareVariable(float3,        W, , )   = { -1.0f, 0.0f, 0.0f };
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , ) = 0.1f;
rtDeclareVariable(rtObject,      top_object, , );
//new variable
rtDeclareVariable(rtObject,      screen_object, , );

rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(int, display_mode,,) = 1;

//new variables
rtBuffer<float3, 2>              output_screen_buffer_f3;
rtBuffer<float4, 2>              output_screen_buffer_f4;

rtBuffer<float3, 2>              output_buffer_f3;
rtBuffer<float4, 2>              output_buffer_f4;
rtDeclareVariable(int,  output_format, , ) = RT_FORMAT_FLOAT4;

//new variables
rtDeclareVariable(uint, screen_sizeX, , );
rtDeclareVariable(uint, screen_sizeY, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

//eye parameters
rtDeclareVariable(float, aperture_radius, , );
rtDeclareVariable(float, focal_scale, , );

rtDeclareVariable(unsigned int, frame_number, , );
rtDeclareVariable(float4, jitter, , );

// #define TIME_VIEW


__device__ __forceinline__ void write_output( float3 c )
{
  if ( output_format == RT_FORMAT_FLOAT4 ) {
    output_buffer_f4[launch_index] = make_float4(c, 1.f);
  }
  else {
    output_buffer_f3[launch_index] = c;
  }
}
__device__ __forceinline__ void write_output_screen(uint2 d, float3 c )
{
  if ( output_format == RT_FORMAT_FLOAT4 ) {
    output_buffer_f4[d] = make_float4(c, 1.f);
  }
  else {
    output_buffer_f3[d] = c;
  }
}
__device__ __forceinline__ float3 read_output()
{
  if ( output_format == RT_FORMAT_FLOAT4 ) {
    return make_float3( output_buffer_f4[launch_index] );
  }
  else {
    return output_buffer_f3[launch_index];
  }
}

__device__ __forceinline__ float3 read_output_screen(uint2 d)
{
  if ( output_format == RT_FORMAT_FLOAT4 ) {
    return make_float3( output_buffer_f4[d] );
  }
  else {
    return output_buffer_f3[d];
  }
}





RT_PROGRAM void dof_camera()
{
#ifdef TIME_VIEW
  clock_t t0 = clock(); 
#endif
  size_t2 screen = output_format == RT_FORMAT_FLOAT4 ? output_buffer_f4.size() : output_buffer_f3.size();

  // pixel sampling
  //fixed size float2 that is never used?
  //float2 l_i = make_float2(512,384);
  float2 pixel_sample = make_float2(launch_index) + make_float2(jitter.x, jitter.y);
  float2 d = pixel_sample / make_float2(screen) * 2.f - 1.f;

  // Calculate ray-viewplane intersection point
  float3 ray_origin = eye; 
  float3 ray_direction = d.x*U + d.y*V + W;
  float3 ray_target = ray_origin + focal_scale * ray_direction;

  // lens sampling
  float2 sample = optix::square_to_disk(make_float2(jitter.z, jitter.w));
  if(useDepthOfField == 1){
  ray_origin = ray_origin + aperture_radius * ( sample.x * normalize( U ) +  sample.y * normalize( V ) );
  ray_direction = normalize(ray_target - ray_origin);
  }
  // shoot ray
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;
  rtTrace(top_object, ray, prd);

  if (frame_number>1)
  {
    float a = 1.0f / (float)frame_number;
    float b = ((float)frame_number - 1.0f) * a;
    const float3 old_color = read_output();
		write_output(a * prd.result + b * old_color);
  }
  else
  {
	    write_output(prd.result);
  }
}
RT_PROGRAM void envmap_miss()
{
	prd_radiance.result = make_float3(0.0,0.0,0.0);
}
RT_PROGRAM void exception()
{
  write_output(make_float3(0.1f,0.5f,0.1f));
}
