#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "distributed_brdf.h"

#include <optixu/optixu_aabb.h>




// hitpoint parameters

// hitpoint parameters
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

//rtTextureSampler<float, 2>     texture_sampler_map;


RT_PROGRAM void any_hit_shadow()
{
  prd_shadow.attenuation = make_float3(0);
  rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{


	  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	  float3 ffnormal  = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
	  //PHONG SHADING
	  //phongShade( kd, ka, ks, ffnormal, phong_exp, reflectivity_n );
	  //BRDF SHADING
	  float3 uvw = texcoord; // testing
	  float3 texCol = make_float3(tex2D(texture_map2,uvw.x*1.5,uvw.y*1.5));
	//  if(uvw.x > 2.0)rtPrintf("LargerX");
	//  if(uvw.y > 2.0)rtPrintf("LargerY");
	//  if(uvw.x < -2.0)rtPrintf("SmallerX");
	//  if(uvw.y < -2.0)rtPrintf("SmallerY");
	//  float x_ = ((uvw.x/2)+0.5f) *256.0f;
	//  float y_ = ((uvw.y/2)+0.5f) *256.0f;
	//  int X = clamp(min(255, (int)x_),0,255);
	//  int Y = clamp(min(255, (int)y_),0,255);
	//  int index = (Y*256)+(X); 
	//  float color = clamp(texture_map[index],0.2f,1.0f);
	//  rtPrintf("X, %d - Y, %d)\n", X, Y);
	 // float3 kas22 = make_float3( color,color,color);
	 float3 textureCol = length(texCol)*make_float3(1.5f,1.5f,1.5f);
	  textureCol += make_float3(0.2,0.2,0.2);
    //textureCol = texcoord; //arp
	  BRDF(ffnormal, textureCol,make_float3(1,1,1));

}